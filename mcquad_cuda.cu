/*
  Problem 1 for Project 3
 
  Program written by Nathan Sanford
  
  Performs Monte Carlo Integration trials in a parallel cuda environment. 
  The integral being approximated is 
                           -inf
			  /
			 |      exp(-x)g(x) dx
			/
                       -0
  where here g(x)=cos(x) but this can be changed in the kernel function mcquad.
  Performs T trials of N samples per trial by throwing each trial onto a 
  processor that draws all the samples for that trial.

  Inputs: Names of 2 text files. The first is an input file which consists 
          of the number of samples per trial and the number of samples. The 
	  second is for an output file to which the result of each trial is
	  written.
  Outputs: Writes the result of T trials to the 
           output file. Additionally, writes the total simulation time to 
	   the terminal at the end of simulation.

  Edit history:
  5/15/2016 Initial draft
  6/3/2016 Added comments
  6/7/2016 Number of trials no longer written to output file
  11/6/2019 Edited filename to differentiate
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
  void init(int T, unsigned int seed, curandState_t* states)

  Initializes the vector of states where each state represents a trial. Uses
  the curand built-in curand_init.

  Inputs:
  T is the number of trials, used as a guard so that only T things are 
  initialized and seed is an integer which represents a seed 
  as in many RNGs.

  Outputs:
  The vector states which is T elements long, is initialized using the built-in
  curand capability curand_init.
*/


__global__ void init(int T, unsigned int seed, hiprandState_t* states) {
  int id=threadIdx.x+blockIdx.x*blockDim.x;
  // we have to initialize the state on each gpu
  if (id<T)
    hiprand_init(seed,id,0,&states[id]);
}

/*
  void mcquad(int T, int N, curandState_t* states, double* results)

  Performs the mcquad trials. Performs T trials by drawing N samples in 
  each trial and calculating the mean. Uses the curand built-in 
  curand_uniform to draw a RN between 0 and 1 and then transform into the 
  exponential distribution. Makes the states local to each processor to 
  speed up the process as the states are updated with the drawing of each sample.
  The function that we're averaging is cos(x) and it is specified within this 
  kernel.

  Inputs:
  T is the number of trials, used as a guard so that only T things are
  initialized, N is the number of samples per trial, and states is the state 
  of the RNG for each trial, where the states are assumed to have been 
  initialized previously.

  Outputs:
  The vector results which is T elements long, where each element is the result
  of a trial.
*/

__global__ void mcquad(int T, int N, hiprandState_t* states, double* results){
  int id=threadIdx.x+blockIdx.x*blockDim.x;
  // ------------------variable declarations-------------------
  double z, y, x; // temporary variables used in drawing samples
  double  mean=0; // mean
  // ----------------------------------------------------------
  // loop through samples
  if (id<T){ // guard so that we only perform T trials
    hiprandState_t localState = states[id];
    for (int n=1; n<=N; n++){
      z=hiprand_uniform(&localState); //uniform RN between 0 and 1
      y=-log(z); //exponential RN
      x=cos(y); //sample
      mean+=x;
    }
    mean*=(1.0/N);
    results[id]=mean;
  }
}

/*
  int main( int argc , char* argv [])

  The main program gets the number of trials and the number of samples and then 
  throws the trials onto the gpu where each processor performs one trial. It 
  chooses the K20c card and then uses the minimal number of blocks to get to 
  T trials by having each thread on each block perform a trial.

  Inputs: should be 3, just function name and input and output text file names

  Outputs: Prints the results of each trial to the
           output file. Additionally prints to the terminal screen the
           time elapsed using the cuda timing stuff. 
*/

int main(int argc, char* argv[])
{  
  // timing the CUDA way  
  hipEvent_t start , stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  int N,T;
  // read parameters for simulation from input file
  if (argc != 3) {
    printf("Incorrect usage: enter the input and output data file names\n");
    return 0;
  }
  FILE* inputfile = fopen(argv[1], "r"); // inputfile only needed temporarily
  if (!inputfile) {
    printf("Unable to open input file\n");
    return 0;
  }
  
  // start reading input data using function fscanf here
  fscanf(inputfile, "%d", &N); // read an integer N for number of samples
  fscanf(inputfile, "%d", &T); // read an integer T for number of trials
  fclose(inputfile);

  // Choose the GPU card
  hipDeviceProp_t prop;
  int dev;
  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.multiProcessorCount = 13;
  hipChooseDevice(&dev,&prop);
  hipSetDevice(dev);

  // Get maximum thread count for the device
  hipGetDeviceProperties(&prop,dev);
  int num_threads = prop.maxThreadsPerBlock;
  int num_blocks = T/num_threads + (T%num_threads ? 1:0);

  // ---------------------------Variable Declarations---------------------
  // for storing results of all the trials
  double* results=(double*)malloc(T*sizeof(double));
  double* dev_results; // results of all trials on device
  hiprandState_t* states;
  // ---------------------------------------------------------------------
  // allocate memory on device
  hipMalloc((void**)&states,T*sizeof(hiprandState_t));
  hipMalloc((void**)&dev_results, T*sizeof(double));

  // set initial seeds
  init<<<num_blocks,num_threads>>>(T,time(NULL),states);

  // do the monte carlo simulations on device
  mcquad<<<num_blocks,num_threads>>>(T,N,states,dev_results);

  // copy output data from device
  hipMemcpy(results,dev_results,T*sizeof(double),hipMemcpyDeviceToHost);

  // write output of trials to file
  FILE *fileid=fopen(argv[2],"w"); // open output file
  fwrite(results,sizeof(double),T,fileid); // write output
  fclose(fileid);    

  // free memory
  free(results);
  hipFree(states);
  hipFree(dev_results);

  // timing the cuda way
  hipEventRecord(stop,0); 
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  // print final execution time to terminal screen
  printf("Cuda time elapsed: %g seconds \n",elapsedTime/1000.0);
  return 0;
}
